#include "hip/hip_runtime.h"

namespace CUDA {
  void
  check_error(std::string msg) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      std::cerr << "CUDA error: "
                << msg << "\n"
                << hipGetErrorString(err) << std::endl;
      exit(EXIT_FAILURE);
    }
  }

  int
  get_num_gpus() {
    int n_gpus;
    hipGetDeviceCount(&n_gpus);
    check_error("trying to get number of available GPUs");
    if (n_gpus == 0) {
      std::cerr << "error: no CUDA-compatible GPUs found."
                << std::endl
                << "       if you are sure to have one,"
                << std::endl
                << "       check your device drivers!"
                << std::endl;
      exit(EXIT_FAILURE);
    } else {
      return n_gpus;
    }
  }

  GPUSettings
  prepare_gpu(int i_gpu
            , unsigned int n_dim
            , std::vector<unsigned int> states
            , const SplitFe& fe
            , const SplitCoord& ref_coords) {
    GPUSettings gpu;
    gpu.id = i_gpu;
    gpu.n_dim = n_dim;
    gpu.states = states;
    hipSetDevice(i_gpu);
    check_error("setting CUDA device");
    for (unsigned int state: states) {
      unsigned int split_size = fe[state].size();
      gpu.split_sizes[state] = split_size;
      //// reserve memory
      hipMalloc((void**) &gpu.fe[state]
               , sizeof(float) * split_size);
      hipMalloc((void**) &gpu.coords[state]
               , sizeof(float) * split_size * gpu.n_dim);
      check_error("device memory allocation");
      //// copy data to device
      hipMemcpy(gpu.fe[state]
               , fe[state].data()
               , sizeof(float) * split_size
               , hipMemcpyHostToDevice);
      check_error("copying of state-splitted free energies");
      // reference coords in 1D array (row-major order)
      std::vector<float> tmp_coords(n_dim * split_size);
      for (unsigned int i=0; i < split_size; ++i) {
        for (unsigned int j=0; j < n_dim; ++j) {
          tmp_coords[i*split_size+j];
        }
      }
      hipMemcpy(gpu.coords[state]
               , tmp_coords.data()
               , sizeof(float) * n_dim * split_size
               , hipMemcpyHostToDevice);
      check_error("copying of state-splitted coordinates");
    }
    return gpu;
  }

  void
  clear_gpu(GPUSettings gpu) {
    hipSetDevice(gpu.id);
    check_error("setting CUDA device");
    for (unsigned int state: gpu.states) {
      hipFree(gpu.fe[state]);
      check_error("freeing memory for free energies");
      hipFree(gpu.coords[state]);
      check_error("freeing memory for coordinates");
    }
  }


  


  //TODO reuse for FE estimate
/*
  Pops
  calculate_populations_per_gpu(const float* coords
                              , std::size_t n_rows
                              , std::size_t n_cols
                              , std::vector<float> radii
                              , std::size_t i_from
                              , std::size_t i_to
                              , int i_gpu) {
    using Clustering::Tools::min_multiplicator;
    unsigned int n_radii = radii.size();
    std::vector<float> rad2(n_radii);
    for (std::size_t i=0; i < n_radii; ++i) {
      rad2[i] = radii[i]*radii[i];
    }
    // GPU setup
    hipSetDevice(i_gpu);
    float* d_coords;
    float* d_rad2;
    unsigned int* d_pops;
    hipMalloc((void**) &d_coords
             , sizeof(float) * n_rows * n_cols);
    hipMalloc((void**) &d_pops
             , sizeof(unsigned int) * n_rows * n_radii);
    hipMalloc((void**) &d_rad2
             , sizeof(float) * n_radii);
    check_error("pop-calc device mallocs");
    hipMemset(d_pops
             , 0
             , sizeof(unsigned int) * n_rows * n_radii);
    check_error("pop-calc memset");
    hipMemcpy(d_coords
             , coords
             , sizeof(float) * n_rows * n_cols
             , hipMemcpyHostToDevice);
    hipMemcpy(d_rad2
             , rad2.data()
             , sizeof(float) * n_radii
             , hipMemcpyHostToDevice);
    check_error("pop-calc mem copies");
    int max_shared_mem;
    hipDeviceGetAttribute(&max_shared_mem
                         , hipDeviceAttributeMaxSharedMemoryPerBlock
                         , i_gpu);
    check_error("getting max shared mem size");
    unsigned int block_size = BSIZE_POPS;
    unsigned int shared_mem = 2 * block_size * n_cols * sizeof(float);
    if (shared_mem > max_shared_mem) {
      std::cerr << "error: max. shared mem per block too small on this GPU.\n"
                << "       either reduce BSIZE_POPS or get a better GPU."
                << std::endl;
      exit(EXIT_FAILURE);
    }
    unsigned int block_rng = min_multiplicator(i_to-i_from, block_size);
    Clustering::logger(std::cout) << "# blocks needed: "
                                  << block_rng << std::endl;
    for (unsigned int i=0; i*block_size < n_rows; ++i) {
      Clustering::Density::CUDA::Kernel::population_count
      <<< block_rng
        , block_size
        , shared_mem >>> (i*block_size
                        , d_coords
                        , n_rows
                        , n_cols
                        , d_rad2
                        , n_radii
                        , d_pops
                        , i_from
                        , i_to);
    }
    hipDeviceSynchronize();
    check_error("after kernel loop");
    // get partial results from GPU
    std::vector<unsigned int> partial_pops(n_rows*n_radii);
    hipMemcpy(partial_pops.data()
             , d_pops
             , sizeof(unsigned int) * n_rows * n_radii
             , hipMemcpyDeviceToHost);
    // sort into resulting pops
    Pops pops;
    for (unsigned int r=0; r < n_radii; ++r) {
      pops[radii[r]].resize(n_rows, 0);
      for (unsigned int i=i_from; i < i_to; ++i) {
        pops[radii[r]][i] = partial_pops[r*n_rows+i];
      }
    }
    hipFree(d_coords);
    hipFree(d_rad2);
    hipFree(d_pops);
    return pops;
  }
*/


  std::pair<unsigned int, float>
  fe_estimate_partial(const std::vector<float>& xs
                    , unsigned int state
                    , unsigned int i_from
                    , unsigned int i_to
                    , GPUSettings& gpu) {
    //TODO
  }

  float
  fe_estimate(const std::vector<float>& xs
            , unsigned int state
            , const std::vector<GPUSettings>& gpus) {
    int n_gpus = gpus.size();
    if (n_gpus == 0) {
      std::cerr << "error: unable to estimate free energies on GPU(s)."
                << std::endl
                << "       no GPUs have been provided."
                << std::endl;
      exit(EXIT_FAILURE);
    }
    unsigned int n_rows = gpus[0].split_sizes[state];
    unsigned int gpu_range = n_rows / n_gpus;
    int i;
    // partial estimates: pair of {#neighbors, sum(FE)}
    std::vector<std::pair<unsigned int, float>> partial_estimates(n_gpus);
    #pragma omp parallel for default(none)\
      private(i)\
      firstprivate(n_gpus,n_rows,n_cols,gpu_range)\
      shared(partial_pops,radii,coords)\
      num_threads(n_gpus)\
      schedule(dynamic,1)
    for (i=0; i < n_gpus; ++i) {
      // use available GPUs in parallel to
      // compute partial estimates
      partial_estimates[i] = fe_estimate_partial(xs
                                               , state
                                               , i*gpu_range
                                               , i == (n_gpus-1)
                                                   ? n_rows
                                                   : (i+1)*gpu_range
                                               , gpus[i]);
    }
    // combine results
    unsigned int n_neighbors = 0;
    float fe_estimate = 0.0f;
    for (auto result: partial_estimates) {
      n_neighbors += result.first;
      fe_estimate += result.second;
    }
    return fe_estimate / ((float) n_neighbors);
  }

}

